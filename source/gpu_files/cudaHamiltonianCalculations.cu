#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

using namespace std;

#include "real_type.h"
#include <iostream>

#include "hostMatrix.hpp"
#include "cudaMatrix.hpp"

#include "cudaHamiltonianCalculations.hpp"

// Possible improvements

////////////////////////////////////////////////////////////////////////////////
// Parallelization helper classes
////////////////////////////////////////////////////////////////////////////////

// The neighbour list setup helper
//
// Note (Thomas):
// For Heisenberg 
// Class sets everything between neighbours and maxneighbours
// to zero since hamiltonian implementation always runs to max neighbours
class CudaHamiltonianCalculations::SetupNeighbourList :
	public CudaParallelizationHelper::Site {
private:
	real *               coup;
	unsigned int *       pos;
	const unsigned int * size;
	unsigned int         mnn;

public:
	SetupNeighbourList(const Exchange &ex) {
		coup  = ex.coupling;
		size  = ex.neighbourCount;
		pos   = ex.neighbourPos;
		mnn   = ex.mnn;
	}

        __device__ void each(unsigned int site) {
		real *         myCoup = &coup[site];
		unsigned int * myPos  = &pos[site];
		unsigned int   mySize = size[site];
		for (unsigned int i = 0; i < mnn; i++) {
			if (i < mySize)
				myPos[i * N]--;
			else {
				myCoup[i * N] = (real)0.0;
				myPos[i * N]  = 0;
			}
		}
	}
};


// The neighbour list setup helper
//
// For Tensorial Exchange 
class CudaHamiltonianCalculations::SetupNeighbourListExchangeTensor :
	public CudaParallelizationHelper::Site {
private:
	real *               tensor;
	unsigned int *       pos;
	const unsigned int * size;
	unsigned int         mnn;

public:
	SetupNeighbourListExchangeTensor(const TensorialExchange &tenEx) {
		tensor  = tenEx.tensor;
		size  = tenEx.neighbourCount;
		pos   = tenEx.neighbourPos;
		mnn   = tenEx.mnn;
	}

        __device__ void each(unsigned int site) {
		//real *         myCoup = &coup[site];
		unsigned int * myPos  = &pos[site];
		unsigned int   mySize = size[site];
		for (unsigned int i = 0; i < mnn; i++) {
			if (i < mySize)
				myPos[i * N]--;
			else {
				//myCoup[i * N] = (real)0.0;
				myPos[i * N]  = 0;

				unsigned int dim1 = 3;
				unsigned int dim2 = 3;
				unsigned int dim3 = mnn;
				unsigned int dim4 = N;

				unsigned int k = i;
				unsigned int l = site;

				// Dimension of the tensorial exchange matrix: (dim1,dim2,dim3,dim4)  <--> (3,3,mnn,N)
				// Calculating the matrix elements of the exchange tensor and setting them to zero:
			    tensor[0 + 3 * (0 + 3 * (k + mnn * l))] = (real)0.0; // i=0,j=0
			    tensor[0 + 3 * (1 + 3 * (k + mnn * l))] = (real)0.0; // i=0,j=1		
			    tensor[0 + 3 * (2 + 3 * (k + mnn * l))] = (real)0.0; // i=0,j=2		
			    tensor[1 + 3 * (0 + 3 * (k + mnn * l))] = (real)0.0; // i=1,j=0		
			    tensor[1 + 3 * (1 + 3 * (k + mnn * l))] = (real)0.0; // i=1,j=1		
			    tensor[1 + 3 * (2 + 3 * (k + mnn * l))] = (real)0.0; // i=1,j=2		
			    tensor[2 + 3 * (0 + 3 * (k + mnn * l))] = (real)0.0; // i=2,j=0		
			    tensor[2 + 3 * (1 + 3 * (k + mnn * l))] = (real)0.0; // i=2,j=1		
			    tensor[2 + 3 * (2 + 3 * (k + mnn * l))] = (real)0.0; // i=2,j=2	
			}
		}
	}
};


// Note (Thomas):
// For DM interaction
// Class sets everything between neighbours and maxneighbours
// to zero since hamiltonian implementation always runs to max neighbours
class CudaHamiltonianCalculations::SetupNeighbourListDM :
	public CudaParallelizationHelper::Site {
private:
	real *               coup;
	unsigned int *       pos;
	const unsigned int * size;
	unsigned int         mnn;

public:
	SetupNeighbourListDM(const DMinteraction & dm) {
		coup  = dm.interaction;
		size  = dm.neighbourCount;
		pos   = dm.neighbourPos;
		mnn   = dm.mnn;
	}

        __device__ void each(unsigned int site) {
		real *         myCoup = &coup[site * 3];
		unsigned int * myPos  = &pos[site];
		unsigned int   mySize = size[site];
		for (unsigned int i = 0; i < mnn; i++) {
			if (i < mySize)
				myPos[i * N]--;
			else {
				myCoup[i * N + 0] = (real)0.0;
				myCoup[i * N + 1] = (real)0.0;
				myCoup[i * N + 2] = (real)0.0;
				myPos[i * N]      = 0;
			}
		}
	}
};

// Note: (Thomas)
// Calculating the magnetic field from various effects
// such as the heisenberg field and DM interactions
// Added DM effect 2014/09/23
class CudaHamiltonianCalculations::HeisgeJij :
	public CudaParallelizationHelper::AtomSiteEnsemble {
private:
	real *               beff;
	const real *         coup;
	const unsigned int * pos;
	const real *         emomM;
	const real *         ext_f;
	unsigned int         mnn;
	const real *         dmcoup;
	const unsigned int * dmpos;
	unsigned int         dmmnn;
public:
	HeisgeJij(real * p1, const real * p2, const real * p3,
			 const Exchange & ex, const DMinteraction & dm) {
		beff   = p1;
		emomM  = p2;
		ext_f  = p3;

		coup   = ex.coupling;
		pos    = ex.neighbourPos;
		mnn    = ex.mnn;

		dmcoup = dm.interaction;
		dmpos  = dm.neighbourPos;
		dmmnn   = dm.mnn; 
	}

        __device__ void each(unsigned int atom, unsigned int site, unsigned int ensemble) {
		// Field
		real x = (real)0.0;
		real y = (real)0.0;
		real z = (real)0.0;

		// Pointers with fixed indices
		const real *         site_coup = &coup[site];
		const unsigned int * site_pos  = &pos[site];
		const real *         my_emomM  = &emomM[ensemble * N * 3];

		const real *         site_dmcoup    = &dmcoup[site];
		const unsigned int * site_dmpos     = &dmpos[site];

		// Exchange term loop
		for (unsigned int i = 0; i < mnn; i++) {
			unsigned int x_offset = site_pos[i * N] * 3; 
			real c = site_coup[i * N];
			x += c * my_emomM[x_offset + 0];
			y += c * my_emomM[x_offset + 1];
			z += c * my_emomM[x_offset + 2];
		}

		// DM interaction, almost no performance impact if dmmnn is 0	
		for (unsigned int i = 0; i < dmmnn; i++) {
			unsigned int x_offset = site_dmpos[i * N] * 3; 
			x += -site_dmcoup[i*N+2]*my_emomM[x_offset+1] + site_dmcoup[i*N+1]*my_emomM[x_offset+2];
			y += -site_dmcoup[i*N+0]*my_emomM[x_offset+2] + site_dmcoup[i*N+2]*my_emomM[x_offset+0];
			z += -site_dmcoup[i*N+1]*my_emomM[x_offset+0] + site_dmcoup[i*N+0]*my_emomM[x_offset+1];
		}

		// Save field
		beff[atom * 3 + 0] = x + ext_f[atom * 3 + 0];
		beff[atom * 3 + 1] = y + ext_f[atom * 3 + 1];
		beff[atom * 3 + 2] = z + ext_f[atom * 3 + 2];
	}
};


class CudaHamiltonianCalculations::HeisJijTensor :
	public CudaParallelizationHelper::AtomSiteEnsemble {
private:
	real *               beff;
	const real *         tensor;
	const unsigned int * pos;
	const unsigned int * size;
	const real *         emomM;
	const real *         ext_f;
	unsigned int         mnn;
public:
	HeisJijTensor(real * p1, const real * p2, const real * p3,
			 const TensorialExchange &tenEx) {
		beff   = p1;
		emomM  = p2;
		ext_f  = p3;

		tensor = tenEx.tensor;
		pos    = tenEx.neighbourPos;
		size   = tenEx.neighbourCount;
		mnn    = tenEx.mnn;
	}

        __device__ void each(unsigned int atom, unsigned int site, unsigned int ensemble) {
		// Field
		real x = (real)0.0;
		real y = (real)0.0;
		real z = (real)0.0;

		// Pointers with fixed indices
		const unsigned int * site_pos  = &pos[site];
		const real *         my_emomM  = &emomM[ensemble * N * 3];



		// Tensorial exchange coupling
		for (unsigned int i = 0; i < mnn; i++) {

			unsigned int x_offset = site_pos[i * N] * 3; 

			unsigned int dim1 = 3;
			unsigned int dim2 = 3;
			unsigned int dim3 = mnn;
			unsigned int dim4 = N;

			unsigned int k = i;
			unsigned int l = site;

			// Dimension of the tensorial exchange matrix: (dim1,dim2,dim3,dim4)  <--> (3,3,mnn,N)
			// Calculating the matrix elements of the exchange tensor:
			//real J11 = tensor[0 + 0 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=0,j=0
			//real J12 = tensor[0 + 1 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=0,j=1
			//real J13 = tensor[0 + 2 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=0,j=2
			//real J21 = tensor[1 + 0 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=1,j=0
			//real J22 = tensor[1 + 1 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=1,j=1
			//real J23 = tensor[1 + 2 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=1,j=2
			//real J31 = tensor[2 + 0 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=2,j=0
			//real J32 = tensor[2 + 1 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=2,j=1
			//real J33 = tensor[2 + 2 * dim1 + k * dim1 * dim2 + l * dim1 * dim2 * dim3]; // i=2,j=2

			//real J11 = tensor[l + k*dim4 + 0*dim4*dim3 + 0*dim4*dim3*dim2]; // i=0,j=0
			//real J12 = tensor[l + k*dim4 + 1*dim4*dim3 + 0*dim4*dim3*dim2]; // i=0,j=1		
			//real J13 = tensor[l + k*dim4 + 2*dim4*dim3 + 0*dim4*dim3*dim2]; // i=0,j=2		
			//real J21 = tensor[l + k*dim4 + 0*dim4*dim3 + 1*dim4*dim3*dim2]; // i=1,j=0		
			//real J22 = tensor[l + k*dim4 + 1*dim4*dim3 + 1*dim4*dim3*dim2]; // i=1,j=1		
			//real J23 = tensor[l + k*dim4 + 2*dim4*dim3 + 1*dim4*dim3*dim2]; // i=1,j=2		
			//real J31 = tensor[l + k*dim4 + 0*dim4*dim3 + 2*dim4*dim3*dim2]; // i=2,j=0		
			//real J32 = tensor[l + k*dim4 + 1*dim4*dim3 + 2*dim4*dim3*dim2]; // i=2,j=1		
			//real J33 = tensor[l + k*dim4 + 2*dim4*dim3 + 2*dim4*dim3*dim2]; // i=2,j=2		

			//const real * J11_temp = &tensor[site]; // i=0,j=0
			//const real * J12_temp = &tensor[site]; // i=0,j=1		
			//const real * J13_temp = &tensor[site]; // i=0,j=2		
			//const real * J21_temp = &tensor[site]; // i=1,j=0		
			//const real * J22_temp = &tensor[site]; // i=1,j=1		
			//const real * J23_temp = &tensor[site]; // i=1,j=2		
			//const real * J31_temp = &tensor[site]; // i=2,j=0		
			//const real * J32_temp = &tensor[site]; // i=2,j=1		
			//const real * J33_temp = &tensor[site]; // i=2,j=2	


			real J11 = tensor[0 + 3 * (0 + 3 * (k + mnn * l))]; // i=0,j=0
			real J12 = tensor[0 + 3 * (1 + 3 * (k + mnn * l))]; // i=0,j=1		
			real J13 = tensor[0 + 3 * (2 + 3 * (k + mnn * l))]; // i=0,j=2		
			real J21 = tensor[1 + 3 * (0 + 3 * (k + mnn * l))]; // i=1,j=0		
			real J22 = tensor[1 + 3 * (1 + 3 * (k + mnn * l))]; // i=1,j=1		
			real J23 = tensor[1 + 3 * (2 + 3 * (k + mnn * l))]; // i=1,j=2		
			real J31 = tensor[2 + 3 * (0 + 3 * (k + mnn * l))]; // i=2,j=0		
			real J32 = tensor[2 + 3 * (1 + 3 * (k + mnn * l))]; // i=2,j=1		
			real J33 = tensor[2 + 3 * (2 + 3 * (k + mnn * l))]; // i=2,j=2	

			//real J11 = J11_temp[i*N + 0]; // i=0,j=0 // 0
			//real J12 = J12_temp[i*N + 1]; // i=0,j=1 // 3		
			//real J13 = J13_temp[i*N + 2]; // i=0,j=2 // 6		
			//real J21 = J21_temp[i*N + 3]; // i=1,j=0 // 1		
			//real J22 = J22_temp[i*N + 4]; // i=1,j=1 // 4		
			//real J23 = J23_temp[i*N + 5]; // i=1,j=2 // 7		
			//real J31 = J31_temp[i*N + 6]; // i=2,j=0 // 2		
			//real J32 = J32_temp[i*N + 7]; // i=2,j=1 // 5		
			//real J33 = J33_temp[i*N + 8]; // i=2,j=2 // 8		

			x += J11 * my_emomM[x_offset + 0] + J12 * my_emomM[x_offset + 1] + J13 * my_emomM[x_offset + 2];
			y += J21 * my_emomM[x_offset + 0] + J22 * my_emomM[x_offset + 1] + J23 * my_emomM[x_offset + 2];
			z += J31 * my_emomM[x_offset + 0] + J32 * my_emomM[x_offset + 1] + J33 * my_emomM[x_offset + 2];
		}

		// Save field
		beff[atom * 3 + 0] = x + ext_f[atom * 3 + 0];
		beff[atom * 3 + 1] = y + ext_f[atom * 3 + 1];
		beff[atom * 3 + 2] = z + ext_f[atom * 3 + 2];
	}
};


class CudaHamiltonianCalculations::HeisgeJijElement :
	public CudaParallelizationHelper::ElementAxisSiteEnsemble {
private:
	real *               beff;
	const real *         coup;
	const unsigned int * pos;
	const unsigned int * size;
	const real *         emomM;
	const real *         ext_f;
	unsigned int         mnn;
public:
	HeisgeJijElement(real * p1, const real * p5, const real * p6, const Exchange & ex) {
		beff   = p1;
		coup   = ex.coupling;
		pos    = ex.neighbourPos;
		size   = ex.neighbourCount;
		emomM  = p5;
		ext_f  = p6;
		mnn    = ex.mnn;
	}

        __device__ void each(unsigned int element, unsigned int axis, unsigned int site, unsigned int ensemble) {
		// Field
		real f = (real)0.0;

		// Pointers with fixed indices
		const real *         site_coup      = &coup[site];
		const unsigned int * site_pos       = &pos[site];
		const real *         ensemble_emomM = &emomM[ensemble * N * 3];

		// Exchange term loop
//		const unsigned int s = size[i];
//		for (int j = 0; j < s; j++) {
		for (unsigned int i = 0; i < mnn; i++) {
			unsigned int offset = site_pos[i * N] * 3;
			f += site_coup[i * N] * ensemble_emomM[offset + axis];
		}

		// Save field
		beff[element] = f + ext_f[element];
	}
};


////////////////////////////////////////////////////////////////////////////////
// Helpers
////////////////////////////////////////////////////////////////////////////////
template<typename T>
static void transpose(T * A, const T * B, size_t M, size_t N) {
	for (size_t y = 0; y < M; ++y)
		for (size_t x = 0; x < N; ++x)
			A[(x * M) + y] = B[(y * N) + x];
}

template <typename T, size_t I, size_t J, size_t K>
static void transpose(hostMatrix<T,2,I,J,K> &A, const hostMatrix<T,2,I,J,K> &B) {
	// Sizes
	size_t M = A.dimension_size(0);
	size_t N = A.dimension_size(1);

	if (B.dimension_size(1) != M || B.dimension_size(0) != N) {
		fprintf(stderr, "Error: illegal matrix transpose\n");
		exit(EXIT_FAILURE);
	}

	transpose(A.get_data(), B.get_data(), M, N);
}

// Function for testing time impact of optimal neighbour alignment
// Will not produce correct results
void alignOptimal(hostMatrix<unsigned int,2> &nlist, bool same) {
	// Sizes
	size_t N   = nlist.dimension_size(0);
	size_t mnn = nlist.dimension_size(1);

	for (size_t m = 0; m < mnn; ++m)
		for (size_t n = 0; n < N; ++n)
			nlist(n,m) = same ? ((m % N) + 1) : (((n + 32 * m) % N) + 1);
}





////////////////////////////////////////////////////////////////////////////////
// Class members
////////////////////////////////////////////////////////////////////////////////

CudaHamiltonianCalculations::CudaHamiltonianCalculations() :
	parallel(CudaParallelizationHelper::def) {
	initiated = false;
}

bool CudaHamiltonianCalculations::initiate(
		const hostMatrix<real,2>         &ncoup, 
		const hostMatrix<unsigned int,2> &nlist,
		const hostMatrix<unsigned int,1> &nlistsize,
		const hostMatrix<real,3,3>       &dm_ncoup, 
		const hostMatrix<unsigned int,2> &dm_nlist,
		const hostMatrix<unsigned int,1> &dm_nlistsize,
		const int 			 do_dm,
		const int do_j_tensor,
		const hostMatrix<real,4,3,3> j_tensor) {

	// Memory access is better if N is multiple of 32
	// (alignment of 128 bytes, see Cuda Best Parctice Guide)
	N      = ncoup.dimension_size(1);    // Number of atoms
	if (N % 32 != 0) {
		printf("Note: Performance is better if the number of atoms is a multiple of 32.\n");
	}


	if (do_j_tensor == 1)
	{
		CudaHamiltonianCalculations::do_j_tensor = true;
		
		tenEx.mnn = ncoup.dimension_size(0);
		tenEx.neighbourCount.clone(nlistsize);
		tenEx.neighbourPos.clone(nlist);
		tenEx.tensor.clone(j_tensor);

		parallel.cudaSiteCall(SetupNeighbourListExchangeTensor(tenEx));

		// Flag
		initiated = true;
		return true;
	}

	//------- Heisenberg Exchange -------//
	ex.mnn    = ncoup.dimension_size(0);    // Max number of neighbours

	// Transposing the matrices will make CUDA calculations faster
	hostMatrix<real,2>         ncoup_t;
	hostMatrix<unsigned int,2> nlist_t;

	ncoup_t.initiate(N,ex.mnn);
	nlist_t.initiate(N,ex.mnn);

	transpose(ncoup_t, ncoup);
	transpose(nlist_t, nlist);

// TEST
//alignOptimal(nlist_t, true);
	//printf("blubb: %f",ex.coupling);

	ex.coupling.clone(ncoup_t);
	ex.neighbourCount.clone(nlistsize);
	ex.neighbourPos.clone(nlist_t);

	// Did we get the memory?
	if (!ex.coupling.has_data()       ||
	    !ex.neighbourCount.has_data() ||
	    !ex.neighbourPos.has_data()) {
		release();
		return false;
	}

	// List setup kernel call
	parallel.cudaSiteCall(SetupNeighbourList(ex));

	//------- DM Interaction -------//
	dm.mnn = 0;
	if (do_dm) {
		dm.mnn = dm_ncoup.dimension_size(0); // Max number of DM neighbours

		dm.interaction.clone(dm_ncoup);
		dm.neighbourCount.clone(dm_nlistsize);
		dm.neighbourPos.clone(dm_nlist);
		
		if (!dm.interaction.has_data()       ||
			!dm.neighbourCount.has_data() ||
			!dm.neighbourPos.has_data()) {
			release();
			return false;
		}
		parallel.cudaSiteCall(SetupNeighbourListDM(dm));
	}

	// Flag
	initiated = true;
	return true;
}


void CudaHamiltonianCalculations::release() {
	ex.coupling.free();
	ex.neighbourCount.free();
	ex.neighbourPos.free();
	dm.interaction.free();
	dm.neighbourCount.free();
	dm.neighbourPos.free();
	initiated = false;
}

void CudaHamiltonianCalculations::heisge(cudaMatrix<real,3,3> &beff, 
		const cudaMatrix<real,3,3> &emomM,
		const cudaMatrix<real,3,3> &external_field) {
	// Kernel call

	if (do_j_tensor == 1)
	{
		parallel.cudaAtomSiteEnsembleCall(HeisJijTensor(beff, emomM, external_field, tenEx));
		return;
	}

	parallel.cudaAtomSiteEnsembleCall(HeisgeJij(beff, emomM, external_field, ex, dm));

	//parallel.cudaElementAxisSiteEnsembleCall(HeisgeJijElement(beff, emomM, external_field, ex));
}
